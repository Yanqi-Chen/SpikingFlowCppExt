#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipblas.h>
using namespace std;

#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
    fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
    exit(-1);}} while(0)
#define CUDA_CALL(X) ERR_NE((X),hipSuccess)
#define CUSPARSE_CALL(X) ERR_NE((X),HIPSPARSE_STATUS_SUCCESS)

template<class T>
struct reCuBuffer
{
T* data = NULL;
int len = 0;
};

template<class T>
void resize(reCuBuffer<T>& buffer, int size)
{
if(size > buffer.len)
{
if(buffer.len > 0)CUDA_CALL( hipFree(buffer.data));
CUDA_CALL( hipMalloc( &(buffer.data), size));
buffer.len = size;
}
}

static reCuBuffer<int>   nnzPerCol_, ColInd_, RowPtr_;
static reCuBuffer<float> csrVal_, tranBuffer_;

void sparse_mm_dense_cusparse_backend(const int & m, const int & n, const int & p, float * dA, float * dB, float * dC)
{   
    // CT = A * BT
    resize(tranBuffer_, m * p * sizeof(float));

    //view_cuda_tensor(A);
    //view_cuda_tensor(B);

    hipsparseHandle_t  handle;
    CUSPARSE_CALL(hipsparseCreate(&handle));
    hipblasHandle_t handle2;
    CUDA_CALL(hipblasCreate(&handle2));

    // transform dense A to csr
    hipsparseMatDescr_t descrX;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrX));

    int total_nnz;
    resize(nnzPerCol_, m * sizeof(int));

    CUSPARSE_CALL(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_COLUMN, n, m, descrX, dA, n, nnzPerCol_.data, &total_nnz));
    resize(csrVal_, total_nnz * sizeof(float));
    resize(ColInd_, total_nnz * sizeof(int));
    resize(RowPtr_, (m+1) * sizeof(int));

    CUSPARSE_CALL(hipsparseSdense2csc(handle, n, m, descrX, dA, n, nnzPerCol_.data, csrVal_.data, ColInd_.data, RowPtr_.data));

    // B * C
    hipsparseMatDescr_t descrA;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CALL(hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO));

    float alpha = 1.0f;
    float beta  = 0.0f;
    CUSPARSE_CALL(hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_TRANSPOSE,
        n,p,m,total_nnz,&alpha,descrA,csrVal_.data,RowPtr_.data, ColInd_.data,dB,p,&beta,tranBuffer_.data,n));

    // C need TRANSPOSE
    CUDA_CALL(hipblasSgeam(handle2, HIPBLAS_OP_T, HIPBLAS_OP_T, p, m, &alpha, tranBuffer_.data, m, &beta, tranBuffer_.data, m, dC, p));
    //view_cuda_tensor(C);

    CUSPARSE_CALL(hipsparseDestroy(handle));
    CUDA_CALL(hipblasDestroy(handle2));
    CUSPARSE_CALL(hipsparseDestroyMatDescr(descrX));
    CUSPARSE_CALL(hipsparseDestroyMatDescr(descrA));
}
