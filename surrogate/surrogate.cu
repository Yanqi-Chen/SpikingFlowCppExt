#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <math.h>
#include <stdio.h>

__global__ void alpha_atan_backward_cuda_kernel(const float* __restrict__ grad_output, const float* __restrict__ x, const float alpha,
    float* __restrict__ grad_x, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
      grad_x[index] = alpha / 2.0f / (1.0f + powf(M_PI_2 * alpha * x[index], 2)) * grad_output[index];
  }
}

__global__ void alpha_sigmoid_backward_cuda_kernel(const float* __restrict__ grad_output, const float* __restrict__ x, const float alpha,
  float* __restrict__ grad_x, const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
      const float sigmoid_ax = 1.0f / (1.0f + __expf(- alpha * x[index]));
      grad_x[index] = grad_output[index] * (1 - sigmoid_ax) * sigmoid_ax * alpha;
  }
}

void alpha_backward_cuda_base(const float* grad_output, const float* x, const float & alpha, float* grad_x, const int & size, 
  void alpha_backward_cuda_kernel(const float* __restrict__, const float* __restrict__, const float, float* __restrict__, const int)
)
{
  const int threads = 1024;
  const int blocks = (size + threads - 1) / threads;
  alpha_backward_cuda_kernel<<<blocks, threads>>>(grad_output, x, alpha, grad_x, size);
}

void alpha_atan_backward_cuda(const float* grad_output, const float* x, const float & alpha, float* grad_x, const int & size)
{
  alpha_backward_cuda_base(grad_output, x, alpha, grad_x, size, alpha_atan_backward_cuda_kernel);
}

void alpha_sigmoid_backward_cuda(const float* grad_output, const float* x, const float & alpha, float* grad_x, const int & size)
{
  alpha_backward_cuda_base(grad_output, x, alpha, grad_x, size, alpha_sigmoid_backward_cuda_kernel);
}
