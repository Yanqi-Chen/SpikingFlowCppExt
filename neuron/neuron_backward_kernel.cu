#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <torch/extension.h>
#include "neuron_def.h"

//LIF bp----------------------------------------------------
__global__ void LIF_backward_cuda_kernel(
    float* __restrict__ grad_x, float* __restrict__ grad_v,
    const float* __restrict__ grad_spike, const float* __restrict__ grad_v_next, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
    const int size,
    const float reciprocal_tau, const float one_sub_reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float grad_h = grad_spike[index] * grad_s_to_h[index] + grad_v_next[index] * grad_v_to_h[index];
    grad_x[index] = grad_h * reciprocal_tau;
    grad_v[index] = grad_h * one_sub_reciprocal_tau;
  }
}

std::vector<at::Tensor> LIF_backward(
  torch::Tensor & grad_spike, torch::Tensor & grad_v_next, torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h,
  const float & reciprocal_tau)
{
  CHECK_TENSOR(grad_spike);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x = torch::zeros_like(grad_spike.data());
  auto grad_v = grad_x.data().clone();
  CHECK_TENSOR(grad_x);
  CHECK_TENSOR(grad_v);
  const int size = grad_spike.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike.get_device()));
  LIF_backward_cuda_kernel<<<blocks, threads>>>(
    grad_x.data_ptr<float>(), grad_v.data_ptr<float>(),
    grad_spike.data_ptr<float>(), grad_v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
    size, reciprocal_tau, 1 - reciprocal_tau
  );
  return {grad_x, grad_v};
}

//LIF bptt----------------------------------------------------

__global__ void LIF_bptt_cuda_kernel(
  float* __restrict__ grad_x_seq, float* __restrict__ grad_v,
  const float* __restrict__ grad_spike_seq, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
  const int neuron_num, const int size,
  const float reciprocal_tau, const float one_sub_reciprocal_tau)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < neuron_num)
  {
    float grad_h;
    for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
    {
      const int mem_index = index + mem_offset;
      grad_h = grad_spike_seq[mem_index] * grad_s_to_h[mem_index] + grad_v[index] * grad_v_to_h[mem_index];
      grad_x_seq[mem_index] = grad_h * reciprocal_tau;
      grad_v[index] = grad_h * one_sub_reciprocal_tau;
    }
  }
}

std::vector<at::Tensor> LIF_bptt(
  torch::Tensor & grad_spike_seq, torch::Tensor & grad_v_next,
  torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h,
  const float & reciprocal_tau)
{
  CHECK_TENSOR(grad_spike_seq);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x_seq = torch::zeros_like(grad_spike_seq.data());
  auto grad_v = grad_v_next.data().clone();
  CHECK_TENSOR(grad_x_seq);
  CHECK_TENSOR(grad_v);
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike_seq.get_device()));
  const int seq_len = grad_spike_seq.size(0);
  const int size = grad_spike_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  LIF_bptt_cuda_kernel<<<blocks, threads>>>(
    grad_x_seq.data_ptr<float>(), grad_v.data_ptr<float>(),
    grad_spike_seq.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
    neuron_num, size,
    reciprocal_tau, 1 - reciprocal_tau
  );
  return {grad_x_seq, grad_v};
}

//IF bp----------------------------------------------------
__global__ void IF_backward_cuda_kernel(
  float* __restrict__ grad_x, float* __restrict__ grad_v,
  const float* __restrict__ grad_spike, const float* __restrict__ grad_v_next, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
  const int size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    const float grad_h = grad_spike[index] * grad_s_to_h[index] + grad_v_next[index] * grad_v_to_h[index];
    grad_x[index] = grad_h;
    grad_v[index] = grad_h;
  }
}

std::vector<at::Tensor> IF_backward(
  torch::Tensor & grad_spike, torch::Tensor & grad_v_next, torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h)
{
  CHECK_TENSOR(grad_spike);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x = torch::zeros_like(grad_spike.data());
  auto grad_v = grad_x.data().clone();
  CHECK_TENSOR(grad_x);
  CHECK_TENSOR(grad_v);
  const int size = grad_spike.numel();
  const int threads = THREADS;
  const int blocks = (size + threads - 1) / threads;
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike.get_device()));
  IF_backward_cuda_kernel<<<blocks, threads>>>(
    grad_x.data_ptr<float>(), grad_v.data_ptr<float>(),
    grad_spike.data_ptr<float>(), grad_v_next.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
    size);
  return {grad_x, grad_v};
}

//IF bptt----------------------------------------------------

__global__ void IF_bptt_cuda_kernel(
float* __restrict__ grad_x_seq, float* __restrict__ grad_v,
const float* __restrict__ grad_spike_seq, const float* __restrict__ grad_s_to_h, const float* __restrict__ grad_v_to_h,
const int neuron_num, const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < neuron_num)
{
  float grad_h;
  for(int mem_offset = size - neuron_num; mem_offset >= 0; mem_offset -= neuron_num)
  {
    const int mem_index = index + mem_offset;
    grad_h = grad_spike_seq[mem_index] * grad_s_to_h[mem_index] + grad_v[index] * grad_v_to_h[mem_index];
    grad_x_seq[mem_index] = grad_h;
    grad_v[index] = grad_h;
  }
}
}

std::vector<at::Tensor> IF_bptt(
  torch::Tensor & grad_spike_seq, torch::Tensor & grad_v_next,
  torch::Tensor & grad_s_to_h, torch::Tensor & grad_v_to_h)
{
  CHECK_TENSOR(grad_spike_seq);
  CHECK_TENSOR(grad_v_next);
  CHECK_TENSOR(grad_s_to_h);
  CHECK_TENSOR(grad_v_to_h);
  auto grad_x_seq = torch::zeros_like(grad_spike_seq.data());
  auto grad_v = grad_v_next.data().clone();
  CHECK_TENSOR(grad_x_seq);
  CHECK_TENSOR(grad_v);
  CHECK_CUDA_OPERATION(hipSetDevice(grad_spike_seq.get_device()));
  const int seq_len = grad_spike_seq.size(0);
  const int size = grad_spike_seq.numel();
  const int threads = THREADS;
  const int neuron_num = size / seq_len;
  const int blocks = (neuron_num + threads - 1) / threads;
  IF_bptt_cuda_kernel<<<blocks, threads>>>(
    grad_x_seq.data_ptr<float>(), grad_v.data_ptr<float>(),
    grad_spike_seq.data_ptr<float>(), grad_s_to_h.data_ptr<float>(), grad_v_to_h.data_ptr<float>(),
    neuron_num, size);
  return {grad_x_seq, grad_v};
}
